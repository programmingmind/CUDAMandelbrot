#include "hip/hip_runtime.h"
#include "datatypes.h"

template <typename VecObject>
__host__
void printList(const VecObject &v) {
   typename VecObject::const_iterator it;
   for (it = v.begin(); it != v.end(); ++it)
      std::cout << *it << " ";
   std::cout << std::endl;
}

inline __host__ __device__
bool numBase2(uint32_t n) {
   return n == 0 || ((n & (n - 1)) == 0);
}

inline __host__ __device__
uint32_t nextBase2(uint32_t n) {
   if (numBase2(n))
      return n;

   uint32_t num;
   for (int i = 0; i < 32; i++)
      if ((num = (1 << i)) >= n)
         return num;
   return num;
}

__host__ __device__
bool topBitsSet(void *data, int len, int numBits) {
   char *d = (char *) data;
   int bytes = numBits / 8, bits = numBits % 8;

   if (numBits > len * 8)
      return true;

   for (int i = 1; i <= bytes; i++)
      if (d[len - i] != 0)
         return true;

   return bits > 0 && (((~((1 << (8 - bits)) - 1)) & d[len - bytes - 1]) != 0);
}

__host__ __device__
uint32_t unsignedAbs(int64_t i) {
   if (i < 0)
      i *= -1;

   return *((uint32_t *) ((void *) &i));
}

template <typename numType>
__host__ __device__
numType max(numType a, numType b) {
   return a > b ? a : b;
}

template <typename numType>
__host__ __device__
numType min(numType a, numType b) {
   return a < b ? a : b;
}

__host__ __device__
int log2(uint32_t n) {
   int i = 0;

   if (!n)
      return -1;

   while (!(n & 1)) {
      n >>= 1;
      ++i;
   }

   return i;
}

__host__ __device__
bool Number::compare(const Number& a, bool lt) {
   int lSize = numBytes >> 2, rSize = a.numBytes >> 2, len = max(lSize, rSize);
   uint32_t *l = (uint32_t *) data, *r = (uint32_t *) a.data;

   for (int i = len - 1; i >= 0; i--) {
      if ((i < lSize ? l[i] : 0) < (i < rSize ? r[i] : 0))
         return lt;
      if ((i < lSize ? l[i] : 0) > (i < rSize ? r[i] : 0))
         return ! lt;
   }
   return false;
}

inline __host__ __device__
Number& Number::copyIn(Number a) {
   // the deallocater frees the pointer so just swap the pointers to handle the memory properly
   void *tmp = data;
   data = a.data;
   a.data = tmp;
   numBytes = a.numBytes;
   return *this;
}

__host__ __device__
int Number::topBytesEmpty() const {
   int len = numBytes;
   unsigned char *ptr = (unsigned char*)data;

   while (len--)
      if (ptr[len])
         return numBytes - (len + 1);

   return numBytes;
}

__host__ __device__
bool Number::nonZero() const {
   uint32_t *ptr = (uint32_t *) data;
   int len = numBytes >> 2;

   while (len--)
      if (ptr[len])
         return true;

   return false;
}

__host__ __device__
splitInfo_t Number::split() const {
   splitInfo_t info;
   info.extra = 0;

   Number tmp(*this);

   int usedBytes = numBytes - topBytesEmpty();
   int num16 = (usedBytes + 1) / 2; // ceil
   if (num16 & 1) {
      ++num16;
      ++info.extra;
      tmp <<= 16;
   }

   int num32 = (tmp.numBytes - tmp.topBytesEmpty() + 3) / 4; // ceil
   if (num32 == 1) {
      tmp <<= 32;
      ++num32;
      info.extra += 2;
   }

   info.len = num32;
   info.data = (uint32_t *) malloc(num32 * 4);
   while (num32--) {
      info.data[num32] = tmp.getLSU32();
      tmp >>= 32;
   }

   return info;
}

__host__ __device__
uint32_t Number::getLSU32() const {
   return *((uint32_t *) data);
}

__host__ __device__
uint32_t Number::getLSU16() const {
   return *((uint16_t *) data);
}

__host__ __device__
Number::Number() {
   onDevice = false;
   numBytes = MIN_BYTES;
   data = malloc(numBytes);
}

__host__ __device__
Number::Number(int bytes) {
   onDevice = false;
   numBytes = nextBase2(max(bytes, MIN_BYTES));
   data = malloc(numBytes);
}

__host__ __device__
Number::Number(const void *bytes, int len) {
   onDevice = false;
   numBytes = nextBase2(len);
   data = malloc(numBytes);
   memset(data, 0, numBytes);
   memcpy(data, bytes, len);
}

__host__ __device__
Number::Number(const Number& num) {
   numBytes = num.numBytes;
   onDevice = num.onDevice;

   if (num.onDevice)
      data = num.data;
   else {
      data = malloc(numBytes);
      memcpy(data, num.data, numBytes);
   }
}

__host__ __device__
Number::~Number() {
   if (! onDevice)
      free(data);
}

__host__ __device__
void Number::resize(int bytes) {
   free(data);
   numBytes = nextBase2(max(bytes, MIN_BYTES));
   data = malloc(numBytes);
   memset(data, 0, numBytes);
}

// returns exponent of first high bit
__host__ __device__
int Number::binlog() const {
   uint32_t *ptr = (uint32_t *) data;
   int len = numBytes >> 2;

   for (int i = 0; i < len; i++)
      if (ptr[i] != 0)
         return i*32 + log2(ptr[i]);

   return 0;
}

__host__ __device__
bool Number::isBase2() const {
   bool base2Seen = false;

   uint32_t *ptr = (uint32_t *) data;
   int len = numBytes >> 2;
   for (int i = 0; i < len; i++) {
      if (ptr[i] != 0) {
         if (numBase2(ptr[i])) {
            if (base2Seen)
               return false;
            base2Seen = true;
         }
         else
            return false;
      }
   }

   return true;
}

__host__ __device__
Number& Number::operator=(const Number& a) {
   if (this == &a)
      return *this;
   onDevice = a.onDevice;

   if (onDevice) {
      numBytes = a.numBytes;
      free(data);
      data = a.data;
   } else {
      if (numBytes < a.numBytes)
         resize(a.numBytes);

      memcpy(data, a.data, a.numBytes);
   }

   return *this;
}

__host__ __device__
Number& Number::operator=(unsigned int a) {
   memset(data, 0, numBytes);
   ((unsigned int *)data)[0] = a;
   return *this;
}

__host__ __device__
Number& Number::operator=(uint64_t a) {
   if (numBytes < 8)
      resize(8);

   memset(data, 0, numBytes);
   ((uint64_t *)data)[0] = a;
   return *this;
}

__host__ __device__
Number Number::operator+(const Number& a) {
   Number n(max(numBytes, a.numBytes));

   int lSize = numBytes >> 2, rSize = a.numBytes >> 2, len = n.numBytes >> 2;
   uint32_t *num1 = ((uint32_t *)data);
   uint32_t *num2 = ((uint32_t *)a.data);

   uint32_t l, r, s;
   char carry = 0;
   for (int i = 0; i < len; i++) {
      l = i < lSize ? num1[i] : 0;
      r = i < rSize ? num2[i] : 0;

      s = l + r + carry;
      carry = ((s < l || s < r) || (carry > 0 && (s == l || s == r))) ? 1 : 0;

      ((uint32_t *)n.data)[i] = s;
   }

   if (carry > 0) {
      Number t(n.numBytes + 1);
      memcpy(t.data, n.data, n.numBytes);
      ((uint32_t *)t.data)[len] = carry;

      return t;
   }

   return n;
}

__host__ __device__
Number Number::operator-(const Number& a) {
   Number n(max(numBytes, a.numBytes));

   uint32_t *num1 = ((uint32_t *)data);
   uint32_t *num2 = ((uint32_t *)a.data);

   int lSize = numBytes >> 2, rSize = a.numBytes >> 2, len = n.numBytes >> 2;

   uint32_t l, r, s;
   char carry = 0;

   for (int i = 0; i < len; i++) {
      l = i < lSize ? num1[i] : 0;
      r = i < rSize ? num2[i] : 0;

      if (r + carry <= l) {
         // normal subtraction
         s = l - r - carry;
         carry = 0;
      } else {
         // l - r == -1 * (r - l)
         s = 1 + ~(r + carry - l);
         carry = 1;
      }

      ((uint32_t *)n.data)[i] = s;
   }

   return n;
}

__host__ __device__
Number Number::operator*(const Number& a) {
   Number p(numBytes + a.numBytes);
   memset(p.data, 0, p.numBytes);

   uint32_t *num1 = ((uint32_t *)data);
   uint32_t *num2 = ((uint32_t *)a.data);

   int lSize = numBytes >> 2, rSize = a.numBytes >> 2;
   uint64_t prod;
   for (int i = 0; i < lSize; i++) {
      for (int j = 0; j < rSize; j++) {
         prod = ((uint64_t) num1[i]) * ((uint64_t) num2[j]);
         Number t(&prod, 8);
         p += (t << ((i + j) * 32));
      }
   }

   p.trim();
   return p;
}

__host__ __device__
Number Number::operator/(const Number& aN) {
   if (! (nonZero() && aN.nonZero())) {
      return *this;
   }

   if (operator==(aN)) {
      Number tmp(4);
      tmp = 1U;
      return tmp;
   }

   if (aN.isBase2())
      return operator>>(aN.binlog());

   splitInfo_t a, c;

   a = aN.split();
   c = split();

   int limit = c.len - a.len + 1;
   int64_t *b = (int64_t *) malloc(limit * 8);

   int64_t tmp = c.data[0] * BASE_SQR + c.data[1];
   b[0] = (tmp / a.data[0]);
   int64_t r = tmp % a.data[0];

   for (int i = 2; i < limit; i++) {
      tmp = r * BASE_SQR + c.data[i];

      for (int j = 1; j < a.len && j < i; j++)
         tmp -= a.data[j] * b[i - j - 1];

      b[i - 1] = (tmp / a.data[0]);
      r = tmp % a.data[0];
   }

   free(a.data);
   free(c.data);

   Number n(b, 8 * limit);
   free(b);
   int shift = 16 * ((a.extra - c.extra) - 2); // * (1 + a.len - c.len));

   if (shift == 0)
      return n;
   else if (shift > 0)
      return n << shift;
   else
      return n >> (-shift);
}

__host__ __device__
Number Number::operator<<(const int a) const {
   int bytes = a / 8;
   int bits = a % 8;

   int clearBytes = topBytesEmpty();
   int overflow = max(0, bytes + (bits > 0 ? 1 : 0) - clearBytes);

   Number t(numBytes + overflow);
   memset(t.data, 0, t.numBytes);

   char *ptr = (char *) t.data;
   if (clearBytes < numBytes)
      memcpy(ptr + bytes, data, numBytes - clearBytes);

   unsigned char mask = (~((1 << (8 - bits)) - 1));
   unsigned char over = 0, tmp;
   for (int i = 0; i < t.numBytes; i++) {
      tmp = ptr[i] & mask;
      ptr[i] <<= bits;
      ptr[i] |= over >> (8 - bits);
      over = tmp;
   }

   return t;
}

__host__ __device__
Number Number::operator>>(const int a) const {
   int bytes = a / 8;
   int bits = a % 8;

   Number t(numBytes - bytes);
   memset(t.data, 0, t.numBytes);

   unsigned char *ptr = (unsigned char *) t.data;
   memcpy(ptr, ((char *) data) + bytes, numBytes - bytes);

   unsigned char mask = (1 << bits) - 1;
   unsigned char under = 0, tmp;
   for (int i = t.numBytes - 1; i >= 0; i--) {
      tmp = ptr[i] & mask;
      ptr[i] >>= bits;
      ptr[i] |= under << (8 - bits);
      under = tmp;
   }

   return t;
}

__host__ __device__
Number Number::operator&(const Number& a) {
   Number n(min(numBytes, a.numBytes));

   memset(n.data, 0, n.numBytes);

   uint32_t *l = (uint32_t *) data, *r = (uint32_t *) a.data, *v = (uint32_t *) n.data;

   for (int i = 0; i*4 < n.numBytes; i++)
      v[i] = l[i] & r[i];

   return n;
}

__host__ __device__
Number Number::operator|(const Number& a) {
   Number n(max(numBytes, a.numBytes));

   int lSize = numBytes >> 2, rSize = a.numBytes >> 2, len = n.numBytes >> 2;
   uint32_t *l = (uint32_t *) data, *r = (uint32_t *) a.data, *v = (uint32_t *) n.data;

   for (int i = 0; i < len; i++)
      v[i] = (i < lSize ? l[i] : 0) | (i < rSize ? r[i] : 0);

   return n;
}

__host__ __device__
Number Number::operator^(const Number& a) {
   Number n(max(numBytes, a.numBytes));

   int lSize = numBytes >> 2, rSize = a.numBytes >> 2, len = n.numBytes >> 2;
   uint32_t *l = (uint32_t *) data, *r = (uint32_t *) a.data, *v = (uint32_t *) n.data;

   for (int i = 0; i < len; i++)
      v[i] = (i < lSize ? l[i] : 0) ^ (i < rSize ? r[i] : 0);

   return n;
}

__host__ __device__
Number& Number::operator+=(const Number& a) {
   return copyIn(operator+(a));
}

__host__ __device__
Number& Number::operator-=(const Number& a) {
   return copyIn(operator-(a));
}

__host__ __device__
Number& Number::operator*=(const Number& a) {
   return copyIn(operator*(a));
}

__host__ __device__
Number& Number::operator/=(const Number& a) {
   return copyIn(operator/(a));
}

__host__ __device__
Number& Number::operator<<=(const int a) {
   return copyIn(operator<<(a));
}

__host__ __device__
Number& Number::operator>>=(const int a) {
   return copyIn(operator>>(a));
}

__host__ __device__
Number& Number::operator&=(const Number& a) {
   return copyIn(operator&(a));
}

__host__ __device__
Number& Number::operator|=(const Number& a) {
   return copyIn(operator|(a));
}

__host__ __device__
Number& Number::operator^=(const Number& a) {
   return copyIn(operator^(a));
}

__host__ __device__
Number& Number::operator&=(const uint32_t a) {
   return copyIn(operator&(a));
}

__host__ __device__
Number& Number::operator|=(const uint32_t a) {
   return copyIn(operator|(a));
}

__host__ __device__
Number& Number::operator^=(const uint32_t a) {
   return copyIn(operator^(a));
}

__host__ __device__
bool Number::operator==(const Number& a) {
   int lSize = numBytes >> 2, rSize = a.numBytes >> 2, len = max(lSize, rSize);
   uint32_t *l = (uint32_t *) data, *r = (uint32_t *) a.data;

   for (int i = 0; i < len; i++)
      if ((i < lSize ? l[i] : 0) != (i < rSize ? r[i] : 0))
         return false;
   return true;
}

__host__ __device__
bool Number::operator!=(const Number& a) {
   return ! operator==(a);
}

__host__ __device__
bool Number::operator>(const Number& a) {
   return compare(a, false);
}

__host__ __device__
bool Number::operator<(const Number& a) {
   return compare(a, true);
}

__host__ __device__
bool Number::operator>=(const Number& a) {
   return ! operator<(a);
}

__host__ __device__
bool Number::operator<=(const Number& a) {
   return ! operator>(a);
}

__host__ __device__
Number Number::operator%(const uint32_t a) {
   uint32_t *d = (uint32_t *) data;
   int len = numBytes >> 2;
   uint64_t mod = 0;

   while (len--)
      mod = ((mod<<32) + d[len]) % a;

   Number n(&mod, 4);

   return n;
}

__host__ __device__
Number Number::operator+(const uint32_t a) {
   Number t(&a, 4);
   return operator+(t);
}

__host__ __device__
Number Number::operator-(const uint32_t a) {
   Number t(&a, 4);
   return operator-(t);
}

__host__ __device__
Number Number::operator*(const uint32_t a) {
   Number t(&a, 4);
   return operator*(t);
}

__host__ __device__
Number Number::operator*(const uint64_t a) {
   Number t(&a, 8);
   return operator*(t);
}

__host__ __device__
Number Number::operator/(const uint32_t a) {
   Number t(&a, 4);
   return operator/(t);
}

__host__ __device__
Number Number::operator+=(const uint32_t a) {
   Number t(&a, 4);
   return operator+=(t);
}

__host__ __device__
Number Number::operator-=(const uint32_t a) {
   Number t(&a, 4);
   return operator-=(t);
}

__host__ __device__
Number Number::operator*=(const uint32_t a) {
   Number t(&a, 4);
   return operator*=(t);
}

__host__ __device__
Number Number::operator/=(const uint32_t a) {
   Number t(&a, 4);
   return operator/=(t);
}

__host__ __device__
Number Number::operator&(const uint32_t a) {
   Number t(&a, 4);
   return operator&(t);
}

__host__ __device__
Number Number::operator|(const uint32_t a) {
   Number t(&a, 4);
   return operator^(t);
}

__host__ __device__
Number Number::operator^(const uint32_t a) {
   Number t(&a, 4);
   return operator^(t);
}

__host__ __device__
bool Number::operator==(const uint32_t a) {
   Number t(&a, 4);
   return operator==(t);
}

__host__ __device__
bool Number::operator!=(const uint32_t a) {
   Number t(&a, 4);
   return operator!=(t);
}

__host__ __device__
bool Number::operator>(const uint32_t a) {
   Number t(&a, 4);
   return operator>(t);
}

__host__ __device__
bool Number::operator<(const uint32_t a) {
   Number t(&a, 4);
   return operator<(t);
}

__host__ __device__
bool Number::operator>=(const uint32_t a) {
   Number t(&a, 4);
   return operator>=(t);
}

__host__ __device__
bool Number::operator<=(const uint32_t a) {
   Number t(&a, 4);
   return operator<=(t);
}

__host__ __device__
void Number::trim() {
   char* ptr = (char *) data;
   int used;

   for (int i = numBytes - 1; i >= 0; i--) {
      if (ptr[i]) {
         used = i + 1;
         break;
      }
   }

   int newBytes = max(nextBase2(used), (uint32_t) MIN_BYTES);
   if (newBytes < numBytes) {
      void *smaller = malloc(newBytes);
      memcpy(smaller, data, newBytes);
      free(data);
      data = smaller;
      numBytes = newBytes;
   }
}

__host__ __device__
void* Number::getData() {
   void *ptr = malloc(numBytes);
   memcpy(ptr, data, numBytes);
   return ptr;
}

__host__ __device__
int Number::getSize() const {
   return numBytes;
}

__host__
std::ostream& operator<<(std::ostream& os, const Number& n) {
   std::ios::fmtflags flags = os.flags();
   int width = os.width();

   int pos = n.numBytes - n.topBytesEmpty();
   if (pos < 1)
      pos = 1;

   unsigned char *ptr = (unsigned char*)n.data;

   os << "0x";
   while (pos--)
      os << std::noshowbase << std::hex << std::setw(2) << std::setfill('0') << (int)ptr[pos];

   os.width(width);
   os.flags(flags);

   return os;
}

__host__ __device__
Number Number::absVal() {
   return *this;
}

__host__ __device__
Number Number::toDevice() const {
#ifdef __HIPCC__
   #ifdef __CUDA__ARCH__
    return *this;
   #else
      if (onDevice)
         return *this;

      Number t(MIN_BYTES);
      free(t.data);

      t.onDevice = true;
      t.numBytes = numBytes;
      hipMalloc(& (t.data), numBytes);
      hipMemcpy(t.data, data, numBytes, hipMemcpyHostToDevice);

      return t;
   #endif
#else
   return *this;
#endif
}

__host__
void Number::deviceFree() {
#ifdef __HIPCC__
   if (onDevice)
      hipFree(data);

   return;
#else
   return;
#endif
}

__host__ __device__
bool Number::isDevice() const {
   return onDevice;
}

__host__ __device__
bool Decimal::compare(const Decimal& a, bool lt) {
   if (negative != a.negative)
      return negative ? lt : !lt;

   if (exponent != a.exponent) {
      Decimal tmp((exponent < a.exponent) ? a : *this);
      tmp.mantissa <<= abs(a.exponent - exponent);
      tmp.exponent = min(exponent, a.exponent);

      return (exponent < a.exponent) ? compare(tmp, lt) : tmp.compare(a, lt);
   }

   if (mantissa < a.mantissa)
      return negative ? !lt : lt;
   else if (mantissa > a.mantissa)
      return negative ? lt : !lt;
   return false;
}

inline __host__ __device__
Decimal& Decimal::copyIn(Decimal d) {
   onDevice = d.onDevice;
   negative = d.negative;
   exponent = d.exponent;
   mantissa = d.mantissa;
   return *this;
}

__host__ __device__
Decimal::Decimal(unsigned int i) : mantissa(4) {
   onDevice = false;
   negative = false;
   exponent = 0;
   mantissa = i;
}

__host__ __device__
Decimal::Decimal(float f) : mantissa(4) {
   onDevice = false;
   union {
      float f;
      uint32_t i;
   } q;
   q.f = f;

   int leading = 1;

   negative = (q.i >> 31) != 0;

   exponent = ((q.i >> 23) & ((1 << 8) - 1));
   if (exponent == 0)
      leading = 0;

   mantissa = (q.i & ((1 << 23) - 1)) | (leading << 23);
   if (mantissa > 0) {
      int low = mantissa.binlog();
      mantissa >>= low;
      exponent-= ((1 << 7) - 1) + 23 - low;
   }
}

__host__ __device__
Decimal::Decimal(double d) : mantissa(8) {
   onDevice = false;
   union {
      double d;
      uint64_t i;
   } q;
   q.d = d;

   uint64_t leading = 1ULL;

   negative = (q.i >> 63) != 0;

   exponent = ((q.i >> 52) & ((1 << 11) - 1));
   if (exponent == 0)
      leading = 0;

   mantissa = (uint64_t) ((q.i & ((1ULL << 52) - 1)) | (leading << 52));
   if (mantissa > 0) {
      int low = mantissa.binlog();
      mantissa >>= low;
      exponent -= ((1 << 10) - 1) + 52 - low;
   }
}

__host__ __device__
Decimal::Decimal(Number &n) : mantissa(n.getSize()) {
   onDevice = false;
   negative = false;
   exponent = 0;
   mantissa = n;
}

__host__ __device__
Decimal::Decimal(const Decimal& d) : mantissa(d.mantissa.getSize()) {
   onDevice = d.onDevice;
   negative = d.negative;
   exponent = d.exponent;
   mantissa = d.mantissa;
}

__host__ __device__
Number Decimal::getMantissa() {
   return mantissa;
}

__host__ __device__
Decimal& Decimal::operator=(const Decimal& a) {
   if (this == &a)
      return *this;

   onDevice = a.onDevice;
   negative = a.negative;
   exponent = a.exponent;
   mantissa = (a.onDevice && !a.mantissa.isDevice()) ? a.mantissa.toDevice() : a.mantissa;
   return *this;
}

__host__ __device__
Decimal Decimal::operator+(const Decimal& a) {
   if (exponent != a.exponent) {
      Decimal tmp((exponent < a.exponent) ? a : *this);
      tmp.mantissa <<= abs(a.exponent - exponent);
      tmp.exponent = min(exponent, a.exponent);

      return (exponent < a.exponent) ? operator+(tmp) : (tmp + a);
   }

   Decimal tmp(a);
   if (negative == a.negative) {
      tmp.mantissa += mantissa;
   } else {
      if (mantissa == a.mantissa)
         return Decimal(0U);
      else if (mantissa < a.mantissa) {
         tmp.negative = a.negative;
         tmp.mantissa -= mantissa;
      }
      else {
         tmp.negative = negative;
         tmp.mantissa = mantissa - a.mantissa;
      }
   }

   return tmp;
}

__host__ __device__
Decimal Decimal::operator-(const Decimal& a) {
   if (exponent != a.exponent) {
      Decimal tmp((exponent < a.exponent) ? a : *this);
      tmp.mantissa <<= abs(a.exponent - exponent);
      tmp.exponent = min(exponent, a.exponent);

      return (exponent < a.exponent) ? operator-(tmp) : (tmp - a);
   }

   Decimal tmp(a);
   if (negative == a.negative) {
      if (mantissa == a.mantissa)
         return Decimal((unsigned int) 0);
      else if (mantissa < a.mantissa) {
         tmp.negative = !negative;
         tmp.mantissa -= mantissa;
      } else {
         tmp.mantissa = mantissa - tmp.mantissa;
      }
   } else {
      tmp.negative = negative;
      tmp.mantissa += mantissa;
   }

   return tmp;
}

__host__ __device__
Decimal Decimal::operator*(const Decimal& a) {
   Decimal tmp(a);

   tmp.negative ^= negative;
   tmp.exponent += exponent;

   if (mantissa.isBase2())
      tmp.exponent += mantissa.binlog();
   else if (a.mantissa.isBase2()) {
      tmp.mantissa = mantissa;
      tmp.exponent += a.mantissa.binlog();
   }
   else
      tmp.mantissa *= mantissa;

   return tmp;
}

__host__ __device__
Decimal Decimal::operator/(const Decimal& a) {
   Decimal tmp(*this);

   int low = a.mantissa.binlog();
   int shift = 32;

   tmp.negative ^= a.negative;
   tmp.exponent -= (a.exponent + low + shift);

   tmp.mantissa <<= shift;
   if (! a.mantissa.isBase2())
      tmp.mantissa /= (a.mantissa >> low);

   return tmp;
}

__host__ __device__
bool Decimal::operator>(const Decimal& a) {
   return compare(a, false);
}

__host__ __device__
bool Decimal::operator<(const Decimal& a) {
   return compare(a, true);
}

__host__ __device__
bool Decimal::operator>=(const Decimal& a) {
   return ! operator<(a);
}

__host__ __device__
bool Decimal::operator<=(const Decimal& a) {
   return ! operator>(a);
}

__host__ __device__
bool Decimal::operator==(const Decimal& a) {
   if (negative != a.negative)
      return false;

   if (exponent != a.exponent) {
      Decimal tmp((exponent < a.exponent) ? a : *this);
      tmp.mantissa <<= abs(a.exponent - exponent);
      tmp.exponent = min(exponent, a.exponent);

      return (exponent < a.exponent) ? operator==(tmp) : (tmp == a);
   }

   return mantissa == a.mantissa;
}

__host__ __device__
bool Decimal::operator>(const uint32_t a) {
   Decimal r(a);
   return operator>(r);
}

__host__ __device__
bool Decimal::operator<(const uint32_t a) {
   Decimal r(a);
   return operator<(r);
}

__host__ __device__
bool Decimal::operator<(const double a) {
   return operator<(Decimal(a));
}

__host__ __device__
bool Decimal::operator>=(const uint32_t a) {
   Decimal r(a);
   return operator>=(r);
}

__host__ __device__
bool Decimal::operator<=(const uint32_t a) {
   Decimal r(a);
   return operator<=(r);
}

__host__ __device__
Decimal& Decimal::operator+=(const Decimal& d) {
   return copyIn(operator+(d));
}

__host__ __device__
Decimal& Decimal::operator/=(const Decimal& d) {
   return copyIn(operator/(d));
}

__host__
std::ostream& operator<<(std::ostream& os, const Decimal& d) {
   os << "{negative: " << d.negative << ", exponent: " << d.exponent << ", mantissa: " << d.mantissa << "}";
   return os;
}

__host__ __device__
Decimal Decimal::absVal() {
   Decimal tmp(*this);
   tmp.negative = false;
   return tmp;
}

__host__
Decimal Decimal::toDevice() const {
#ifdef __HIPCC__
   if (onDevice)
      return *this;

   Decimal d(*this);
   d.onDevice = true;
   d.mantissa = d.mantissa.toDevice();

   return d;
#else
   return *this;
#endif
}

__host__
void Decimal::deviceFree() {
#ifdef __HIPCC__
   mantissa.deviceFree();
#else
   return;
#endif
}
