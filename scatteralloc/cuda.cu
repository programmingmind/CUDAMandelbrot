#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "common.h"

#define cudaSafe(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t code, char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"cudaAssert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort)
         exit(code);
   }
}

__global__ void iterate(data_t startX, data_t startY, data_t resolution, uint32_t *iters) {
   unsigned int yNdx = blockIdx.y * blockDim.y + threadIdx.y;
   unsigned int xNdx = blockIdx.x * blockDim.x + threadIdx.x;

   startX.clearDevice();
   startY.clearDevice();
   resolution.clearDevice();

   if (xNdx < WIDTH && yNdx < HEIGHT) {
      uint32_t it=0;	
      data_t x0 = startX + ((resolution * xNdx) / WIDTH);
      data_t y0 = startY + ((resolution * yNdx) / HEIGHT);
      data_t x = x0, y = y0;
      data_t xSqr((unsigned int) 0), ySqr((unsigned int) 0);

      while (((xSqr = x*x) + (ySqr = y*y) <= 4) && (it < MAX)) {
         y = x*y*((unsigned int) 2) + y0;
         x = xSqr - ySqr + x0;
         it++;
      }

      iters[yNdx * WIDTH + xNdx] =  it;
   }
}

void Mandelbrot(data_t x, data_t y, data_t resolution, uint32_t *iters, bool first) {
   if (first) {
      initializeHeap();
   }

   uint32_t *cuda;
   int size = WIDTH * HEIGHT * sizeof(uint32_t);
   cudaSafe(hipMalloc(&cuda, size));

   dim3 dimGrid(1 + (WIDTH - 1)/BLOCK_LEN, 1 + (HEIGHT - 1)/BLOCK_LEN);
   dim3 dimBlock(BLOCK_LEN, BLOCK_LEN);

   data_t dX = x.toDevice();
   data_t dY = y.toDevice();
   data_t dR = resolution.toDevice();

   hipDeviceSetLimit(hipLimitStackSize, 1024*2);
   hipDeviceSetLimit(hipLimitMallocHeapSize, 536870912);
   iterate<<<dimGrid, dimBlock>>>(dX, dY, dR, cuda);
   cudaSafe(hipPeekAtLastError());
   cudaSafe(hipDeviceSynchronize());

   dX.deviceFree();
   dY.deviceFree();
   dR.deviceFree();

   cudaSafe(hipMemcpy(iters, cuda, size, hipMemcpyDeviceToHost));
   cudaSafe(hipFree(cuda));
}
