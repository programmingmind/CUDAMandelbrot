
#include <hip/hip_runtime.h>
#include <inttypes.h>

#include <algorithm>
#include <cmath>
#include <ctime>
#include <fstream>

using namespace std;

#define WIDTH 480
#define HEIGHT WIDTH
#define INITIAL_RESOLUTION 2.0

#define MAX 65536
#define DEPTH 15

#define BLOCK_LEN 32

#define STD_DEV_RADIUS 5
#define RANDOM_POOL_SIZE (1 << 2)

typedef struct {
   double variance;
	double mean;
   int xNdx;
	int yNdx;
} StdDevInfo_t;

inline uint32_t getColor(uint32_t it) {
   unsigned char R = (it>>2)&1 | (it>>4)&2 | (it>>6)&4 | (it>>8)&8 | (it>>10)&16;
   unsigned char G = (it>>1)&1 | (it>>3)&2 | (it>>5)&4 | (it>>7)&8 | (it>>9)&16;
   unsigned char B = it&1 | (it>>2)&2 | (it>>4)&4 | (it>>6)&8 | (it>>8)&16 | (it>>10)&32;
   return B<<2 | G<<11 | R<<19;
}

__global__ void iterate(double startX, double startY, double resolution, uint32_t *iters) {
   int yNdx = blockIdx.y * blockDim.y + threadIdx.y;
	int xNdx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (xNdx < WIDTH && yNdx < HEIGHT) {
      uint32_t it=0;	
      double x0 = startX + (xNdx * resolution / WIDTH);
      double y0 = startY + (yNdx * resolution / HEIGHT);
		double x = x0, y = y0;
      double xSqr, ySqr;
	   
	   while (((xSqr = x*x) + (ySqr = y*y) <= 4) && (it < MAX)) {
         y = 2*x*y + y0;
         x = xSqr - ySqr + x0;
         it++;
	   }
	   
      iters[yNdx * WIDTH + xNdx] =  it;
	}
}

void saveImage(char *name, uint32_t *iters) {
   char bfType1=0x42;
   char bfType2=0x4D;
   char bfSize[4]={0x00,0x00,0x00,0x00}; 
   char bfReserved1[2]={0x00,0x00};
   char bfReserved2[2]={0x00,0x00};
   char bfOffset[4]={0x36,0x00,0x00,0x00};
   char biSize[4]={0x28,0x00,0x00,0x00};
	
   char biWidth[4] = {char (WIDTH),
	                   char (WIDTH >> 8),
							 char (WIDTH >> 16),
							 char (WIDTH >> 24)};
   char biHeight[4] = {char (HEIGHT),
	                    char (HEIGHT >> 8),
							  char (HEIGHT >> 16),
							  char (HEIGHT >> 24)};
							 
   char biPlanes[2]={0x01,0x00};
   char biBitCount[2]={0x18,0x00};
   char biCompression[4]={0x00,0x00,0x00,0x00};
   char biSizeImage[4]={0x00,0x00,0x00,0x00};
   char biXPelsPerMeter[4]={0x00,0x00,0x00,0x00};
   char biYPelsPerMeter[4]={0x00,0x00,0x00,0x00};
   char biClrUsed[4]={0x00,0x00,0x00,0x00};
   char biClrImportant[4]={0x00,0x00,0x00,0x00};

   fstream image;
   image.open (name, fstream::binary | fstream::out);
	
   image.put(bfType1);
   image.put(bfType2);
   for(int i=0;i<4;i++){image.put(bfSize[i]);}
   for(int i=0;i<2;i++){image.put(bfReserved1[i]);}
   for(int i=0;i<2;i++){image.put(bfReserved2[i]);}
   for(int i=0;i<4;i++){image.put(bfOffset[i]);}
   for(int i=0;i<4;i++){image.put(biSize[i]);}
   for(int i=0;i<4;i++){image.put(biWidth[i]);}
   for(int i=0;i<4;i++){image.put(biHeight[i]);}
   for(int i=0;i<2;i++){image.put(biPlanes[i]);}
   for(int i=0;i<2;i++){image.put(biBitCount[i]);}
   for(int i=0;i<4;i++){image.put(biCompression[i]);}
   for(int i=0;i<4;i++){image.put(biSizeImage[i]);}
   for(int i=0;i<4;i++){image.put(biXPelsPerMeter[i]);}
   for(int i=0;i<4;i++){image.put(biYPelsPerMeter[i]);}
   for(int i=0;i<4;i++){image.put(biClrUsed[i]);}
   for(int i=0;i<4;i++){image.put(biClrImportant[i]);}

	int offset = 4 - ((WIDTH * 3) % 4);
	if (offset == 4)
	   offset = 0;
	
   for (int i = 0; i < HEIGHT; i++) {
      for (int j = 0; j < WIDTH; j++) {
		   uint32_t color = getColor(iters[i * WIDTH + j]);
         image.put(char(color >> 16));
         image.put(char(color >> 8));
         image.put(char(color));
      }
		
      for (int j = 0; j < offset; j++)
         image.put(0);
   }
   
	image.close();
}

inline bool BetterZoom(double oMean, double oVar, double nMean, double nVar) {
   return nVar > oVar;
}

double Variance(uint32_t iters[], double mean, uint32_t count) {
   if (count == 0)
      return 0.0;
	
   double sqrSum = 0.0;
   for (int i = 0; i < count; i++)
      sqrSum = pow(mean - (double)iters[i], 2);
	
   return (sqrSum/(count-1));
}

void insertSorted(StdDevInfo_t stdDevs[], int *varCount, uint32_t iters[], int count, int xNdx, int yNdx) {
   if (count == 0)
	   return;
	
	uint32_t sum = 0;
	int ndx = *varCount;
	double mean, variance;
	
	for (int i = 0; i < count; i++)
	   sum += iters[i];
	
	mean = (double) sum / (double) count;
	variance = Variance(iters, sum, count);
	
	while (ndx > 0 && BetterZoom(stdDevs[ndx - 1].mean, stdDevs[ndx - 1].variance, mean, variance)) {
	   if (ndx < RANDOM_POOL_SIZE)
	      stdDevs[ndx] = stdDevs[ndx - 1];
	   ndx--;
	}
	
	if (ndx < RANDOM_POOL_SIZE) {
	   if (*varCount < RANDOM_POOL_SIZE)
		   ++*varCount;
		
		stdDevs[ndx].variance = variance;
		stdDevs[ndx].mean = mean;
		stdDevs[ndx].xNdx = xNdx;
		stdDevs[ndx].yNdx = yNdx;
	}
}

void Mandelbrot(double x, double y, double resolution, uint32_t *iters) {
   uint32_t *cuda;
	int size = WIDTH * HEIGHT * sizeof(uint32_t);
	hipMalloc(&cuda, size);
	
	dim3 dimGrid(1 + (WIDTH - 1)/BLOCK_LEN, 1 + (HEIGHT - 1)/BLOCK_LEN);
	dim3 dimBlock(BLOCK_LEN, BLOCK_LEN);
	
	iterate<<<dimGrid, dimBlock>>>(x, y, resolution, cuda);
	
	hipMemcpy(iters, cuda, size, hipMemcpyDeviceToHost);
}

void findPath(uint32_t *iters, double *startX, double *startY, double *resolution) {
   int count;
	uint32_t subIter[(2*STD_DEV_RADIUS+1)*(2*STD_DEV_RADIUS+1)];
	
	StdDevInfo_t stdDevs[RANDOM_POOL_SIZE];
	int varCount = 0;
	
	for (int i = 0; i < HEIGHT; i++) {
	   for (int j = 0; j < WIDTH; j++) {
		   count = 0;
			
			for (int k = max(0, i - STD_DEV_RADIUS); k <= min(HEIGHT - 1, i + STD_DEV_RADIUS); k++)
			   for (int l = max(0, j - STD_DEV_RADIUS); l <= min(WIDTH - 1, j + STD_DEV_RADIUS); l++)
				   subIter[count++] = getColor(iters[k * WIDTH + l]);
			
			insertSorted(stdDevs, &varCount, subIter, count, j, i);
		}
	}
	
	int path = clock() & (RANDOM_POOL_SIZE - 1);
	
	*startX += stdDevs[path].xNdx * *resolution / WIDTH;
	*startY += stdDevs[path].yNdx * *resolution / HEIGHT;
	
	(*resolution) /= 2.0;
}

int main(int argc, char* argv[]) {
   double startX = -1.50;
	double startY = -1.00;
	double resolution = INITIAL_RESOLUTION;
	
	uint32_t iters[WIDTH * HEIGHT];

	char file[] = "a.bmp";
   for (int i = 0; i < DEPTH; i++, file[0]++) {
	   Mandelbrot(startX, startY, resolution, iters);
		saveImage(file, iters);
		findPath(iters, &startX, &startY, &resolution);
	}
	
   return 0;
}
